#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "matrixOperations.cu"

using namespace std;

int main() {

    srand(time(nullptr));

    int n = 5000;

    int** matrizA = crearMatriz(n);
    int** matrizB = crearMatriz(n);

    matrizResultante(matrizA, matrizB, n);


    int** mtxCudaA = crearMatriz(n);
    int** mtxCudaB = crearMatriz(n);

    hipMalloc(&mtxCudaA, n * n * sizeof(int));
    hipMalloc(&mtxCudaB, n * n * sizeof(int));

    hipMemcpy(mtxCudaA, matrizA, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(mtxCudaB, matrizB, n * n * sizeof(int), hipMemcpyHostToDevice);

    clock_t init, end;
    double timeExec;

    init = clock();
    matrixMultiplication <<< 1, 256 >>> (mtxCudaA, mtxCudaB, n);
    hipDeviceSynchronize();
    end = clock();

    timeExec = ((double)(end - init)) / CLOCKS_PER_SEC;

    cout << "Tiempo total de procesamiento en CUDA: " << timeExec << endl;

    return 0;
}

