#include <iostream>
#include <stdlib.h>
#include <vector>
#include <ctime>
#include <hip/hip_runtime.h>


using namespace std;

vector<int> generateVector(int upLim, int downLim, int n)
{
    vector<int> vector;

    for (int i = 0; i < n; i++)
    {
        vector.push_back(downLim + rand() % (upLim - downLim));
    }

    cout << "Vector creado: " << endl;

    return vector;
}

void matrizResultante(int** matriz1, int** matriz2, int n)
{
    clock_t init, end;
    double timeExec;

    int** resultado = new int*[n];

    init = clock();

    for (int i = 0; i < n; ++i)
    {
        resultado[i] = new int[n];
    }
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            resultado[i][j] = 0;
            for (int k = 0; k < n; ++k)
            {
                resultado[i][j] += matriz1[i][k] * matriz2[k][j];
            }
        }
    }

    end = clock();

    timeExec = ((double)(end - init)) / CLOCKS_PER_SEC;

    cout << "Tiempo total de procesamiento: " << timeExec << endl;
}

// Función para crear una matriz n x n con números aleatorios
int **crearMatriz(int n)
{
    // Reservar memoria para una matriz dinámica de n x n
    int **matriz = new int *[n];
    for (int i = 0; i < n; ++i)
    {
        matriz[i] = new int[n];
    }

    // Llenar la matriz con números aleatorios
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            matriz[i][j] = std::rand() % 100; // Números aleatorios entre 0 y 99
        }
    }

    cout << "Matriz creada: " << endl;

    return matriz;
}


__global__ void matrixMultiplication(int** A, int** B, int n) {

    // Reservar memoria para una matriz dinámica de n x n
    int **matriz = new int *[n];
    for (int i = 0; i < n; ++i)
    {
        matriz[i] = new int[n];
    }

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int i = 0; i < n; i++) {
            sum += A[row][i] * B[i][col];
        }
        matriz[row][col] = sum;
    }
}